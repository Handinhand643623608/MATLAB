#include "hip/hip_runtime.h"


/* DEPENDENCIES */
#include "mex.h"
#include "gpu/mxGPUArray.h"



/* MACROS */
#define Error mexErrMsgIdAndTxt;




/* FUNCTION PROTOTYPES */
__global__ void CalculateTailValues(const double *realData, const double *nullData, const int lenNull, double* pvals);



/* ENTRY POINT */
void mexFunction(const int numOutputs, mxArray *pOutputs[], const int numInputs, const mxArray* pInputs[])
{

    // Declare variables
    mxGPUArray* pvalsGPU;
    
    
    
    // Error check
//     if (numInputs != 2) { Error("EmpiricalDistribution:NumInputs", "Two inputs must be provided to this function."); }
//     if (numOutputs != 1) { Error("EmpiricalDistribution:NumOutputs", "Only one output may be requested from this function."); }
    
    
    
    // Initialize the GPU API
    mxInitGPU();
    
    
    
    // Get the real & null data sets that were inputted
    const mxArray *realDataCPU = (mxArray *)mxGetPr(pInputs[0]);
    const mxArray *nullDataCPU = (mxArray *)mxGetPr(pInputs[1]);
    const int lenReal = mxGetN(pInputs[0]);
    const int lenNull = mxGetN(pInputs[1]);
    
    
    
    // Make read-only copies of inputted data sets on the GPU
    const mxGPUArray *realDataGPU = mxGPUCreateFromMxArray(realDataCPU);
    const mxGPUArray *nullDataGPU = mxGPUCreateFromMxArray(nullDataCPU);
    
    
    mwSize szPVals[] = { 2, lenReal };
    
    
    pvalsGPU = mxGPUCreateGPUArray(2, szPVals, mxGPUGetClassID(realDataGPU), mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    
    
    
    CalculateTailValues<<<1, 1000>>>((double *)realDataGPU, (double *)nullDataGPU, lenNull, (double *)pvalsGPU);
    
    
    
    pOutputs[0] = mxGPUCreateMxArrayOnCPU(pvalsGPU);
    
    // Clear data off of the GPU
    mxGPUDestroyGPUArray(realDataGPU);
    mxGPUDestroyGPUArray(nullDataGPU);
    mxGPUDestroyGPUArray(pvalsGPU);
}



/* NESTED FUNCTIONS */
__global__ void CalculateTailValues(const double *realData, const double *nullData, const int lenNull, double *pvals)
{
    int a;
    const int i = threadIdx.x;
    int sumNullLower = 0;
    
    for (a = 0; a < lenNull; a++) { if (nullData[a] <= realData[i]) { sumNullLower++; } }
    
    pvals[2*i] = (double)sumNullLower / (double)lenNull;
}
        
    



/* RESULTS */
// NullData [1 5*length(RealData)]
//
// RealData [1 1000] 
// RealData [1 10000]   --> 1.4219s, 1.4220s