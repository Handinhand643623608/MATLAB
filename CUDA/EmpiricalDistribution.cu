#include "hip/hip_runtime.h"
/* EMPIRICALDISTRIBUTION - */

/* CHANGELOG
 * Written by Josh Grooms on 20140903
 */

/* TODOS
 * - Dynamically calculate the number of blocks/threads to call the kernel with
 * - Implement (optional?) conversion of double arrays to float arrays (better for GeForce GPUs)
 */

/* DEPENDENCIES */
#include "mex.h"
#include "gpu/mxGPUArray.h"



/* MACROS */
#define Error(id, txt) mexErrMsgIdAndTxt(id, txt);



/* FUNCTION PROTOTYPES */
__global__ void CalculateTailValues(const double *realData, const double *nullData, const int lenNull, double* pvals);



/* MEX ENTRY POINT */
void mexFunction(const int numOutputs, mxArray *pOutputs[], const int numInputs, const mxArray* pInputs[])
{

    // Declare variables
    mxGPUArray* pvalsGPU;
    
    // Error check
    if (numInputs != 2) { Error("EmpiricalDistribution:NumInputs", "Two inputs must be provided to this function."); }
    if (numOutputs != 1) { Error("EmpiricalDistribution:NumOutputs", "Only one output may be requested from this function."); }
    
    // Initialize the GPU API
    mxInitGPU();
    
    // Get the real & null data sets that were inputted
    const mxArray *realDataCPU = (mxArray *)mxGetPr(pInputs[0]);
    const mxArray *nullDataCPU = (mxArray *)mxGetPr(pInputs[1]);
    const int lenReal = mxGetN(pInputs[0]);
    const int lenNull = mxGetN(pInputs[1]);
    
    // Make read-only copies of inputted data sets on the GPU
    const mxGPUArray *realDataGPU = mxGPUCreateFromMxArray(realDataCPU);
    const mxGPUArray *nullDataGPU = mxGPUCreateFromMxArray(nullDataCPU);
    
    // Initialize the p-value output array on the GPU
    mwSize szPVals[] = { 2, lenReal };
    pvalsGPU = mxGPUCreateGPUArray(2, szPVals, mxGPUGetClassID(realDataGPU), mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    
    // Call the CUDA kernel
    CalculateTailValues<<<1, 1000>>>((double *)realDataGPU, (double *)nullDataGPU, lenNull, (double *)pvalsGPU);
    
    // Transfer GPU p-value array back to the host
    pOutputs[0] = mxGPUCreateMxArrayOnCPU(pvalsGPU);
    
    // Clear data off of the GPU
    mxGPUDestroyGPUArray(realDataGPU);
    mxGPUDestroyGPUArray(nullDataGPU);
    mxGPUDestroyGPUArray(pvalsGPU);
}



/* NESTED FUNCTIONS */
__global__ void CalculateTailValues(const double *realData, const double *nullData, const int lenNull, double *pvals)
{
    int a;
    const int i = threadIdx.x;
    int sumNullLower = 0;
    
    for (a = 0; a < lenNull; a++) { if (nullData[a] <= realData[i]) { sumNullLower++; } }
    
    pvals[2*i] = (double)sumNullLower / (double)lenNull;
}